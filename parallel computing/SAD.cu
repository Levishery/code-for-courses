#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""

#include <stdio.h>
#include<stdlib.h>
#include<math.h>

#define WIDTH 512
#define LIMIT 40
#define WINSIZE 12

#define BLOCK_SIZE 512

int** sad(int view1[WIDTH][WIDTH], int view5[WIDTH][WIDTH]);

__global__ void sad_gpu(int* dis, int* v1, int* v5, int width){
	//每行共享存储
	__shared__ int v1_c[WINSIZE][WIDTH];
	__shared__ int v5_c[WINSIZE][WIDTH];
	//同一行用一个block中的线程计算
	int bi = blockIdx.x;
//	printf("id: %d", bi);
	//一行中的每个像素对应视差用不同thread计算
	int tj = threadIdx.x;
	//每个线程取它对应的像素列
	for(int k = 0; k < WINSIZE; k++){
		v1_c[k][tj] = v1[(bi+k)*WIDTH + tj];
		v5_c[k][tj] = v5[(bi+k)*WIDTH + tj];
	}
	__syncthreads();	//等所有都取完再计算
	//每不越界的线程计算滑窗像素和，找出最接近的
	if(tj < WIDTH-WINSIZE){
		int min = 2550000;
		int min_index = 0;
		for(int k = 0; k < LIMIT; k++){	//滑窗
			unsigned int result = 0;
			for(int m = 0; m < WINSIZE; m++){
				for(int l = 0; l < WINSIZE; l++ ){
					result = __sad(v5_c[m][tj+l], v1_c[m][tj+k+l], result);
				}
			}
			if(result < min){
				min = result;
				min_index = k;
			}
		}
//		printf("%d	",min_index);
		dis[bi*WIDTH+tj] = min_index; 		
	}
}

__global__ void sad_gpu_without_sm(int* dis, int* v1, int* v5, int width){
        int bi = blockIdx.x;
        int tj = threadIdx.x;
        if(tj < WIDTH-WINSIZE){
                int min = 2550000;
                int min_index = 0;
                for(int k = 0; k < LIMIT; k++){ //?~Q窪
                        unsigned int result = 0;
                        for(int m = 0; m < WINSIZE; m++){
                                for(int l = 0; l < WINSIZE; l++ ){
                                        result = __sad(v5[(bi+m)*WIDTH+tj+l], v1[(bi+m)*WIDTH+tj+k+l], result);
                                }
                        }
                        if(result < min){
                                min = result;
                                min_index = k;
                        }
                }
                dis[bi*WIDTH+tj] = min_index;
        }
}

__global__ void sad_gpu2(int* dis, int* v1, int* v5, int width){
        //每行共享存储
        __shared__ int v1_c[WIDTH];
        __shared__ int v5_c[WIDTH];
        //同一行用一个block中的线程计算
        int bi = blockIdx.x;
//      printf("id: %d", bi);
        //一行中的每个像素对应视差用不同thread计算
        int tj = threadIdx.x;
        //每个线程取它对应的像素列计算和
        for(int k = 0; k < WINSIZE; k++){
                v1_c[tj] += v1[(bi+k)*WIDTH + tj];
                v5_c[tj] += v5[(bi+k)*WIDTH + tj];
        }
        __syncthreads();        //等所有都取完再计算
        //每不越界的线程计算滑窗像素和，找出最接近的
        if(tj < WIDTH-WINSIZE){
                int min = 2550000;
                int min_index = 0; 
                for(int k = 0; k < LIMIT; k++){ //滑窗
                        unsigned int result = 0;
                                for(int l = 0; l < WINSIZE; l++ ){
                                        result = __sad(v5_c[tj+l], v1_c[tj+k+l], result);
                                }
                        if(result < min){
                                min = result;
                                min_index = k;
                        }
                }
//              printf("%d      ",min_index);
                dis[bi*WIDTH+tj] = min_index;
        }
}

	
int main()
{
	FILE *view1 = NULL;
	FILE *dis = NULL;
	view1 = fopen("view1.txt", "r");
	static int view1_[WIDTH][WIDTH];
	int i, j;
	char buff;
	for (i = 0; i < WIDTH; i++) {
		for (j = 0; j < WIDTH; j++)
			fscanf(view1, "%d " ,&view1_[i][j]);
		fscanf(view1, "%c", &buff);
	}
	//printf("%d %d\n", view1_[0][0], view1_[511][511]);
	fclose(view1);
	FILE *view5 = NULL;
	view5 = fopen("view5.txt", "r");
	static int view5_[WIDTH][WIDTH];
	for (i = 0; i < WIDTH; i++) {
		for (j = 0; j < WIDTH; j++)
			fscanf(view5, "%d ", &view5_[i][j]);
		fscanf(view5, "%c", &buff);
	}
	//printf("%d %d\n", view5_[0][0], view5_[511][511]);
	fclose(view5);
	int **disp;
	clock_t startTime = clock();	//计时
	disp = sad(view1_, view5_);
	clock_t endTime = clock();
        double CPUtime = (double)(endTime - startTime) / CLOCKS_PER_SEC;
        printf("CPU:%f\n", CPUtime);
	dis = fopen("disp.txt","w");
	for (i = 0; i < WIDTH; i++) {
                for (j = 0; j < WIDTH; j++)
                        fprintf(dis, "%d " ,disp[i][j]);
                fprintf(view1, "\n");
//		printf("write %d\n", i);
        }
	printf("write done\n");
	fclose(dis);
	
	//并行
	//把数据做成一维
	unsigned int size_im = sizeof(int)*WIDTH*WIDTH;
	int *h_1 = (int*)malloc(sizeof(int)*WIDTH*WIDTH);
	int *h_5 = (int*)malloc(sizeof(int)*WIDTH*WIDTH);
	for(i = 0; i < WIDTH*WIDTH; i++){
		h_1[i] = view1_[i/WIDTH][i-(i/WIDTH)*WIDTH];
		h_5[i] = view5_[i/WIDTH][i-(i/WIDTH)*WIDTH];
	}
	//分配设备内存
//	startTime = clock();
	int* d_1;
	hipMalloc((void**)&d_1, size_im);
	int* d_5;
	hipMalloc((void**)&d_5, size_im);
	//拷贝数据到设备
	hipMemcpy(d_1, h_1, size_im, hipMemcpyHostToDevice);
	hipMemcpy(d_5, h_5, size_im, hipMemcpyHostToDevice);
	//分配结果所需空间
	int* d_dis;
	hipMalloc((void**)&d_dis, size_im);
	int* h_dis = (int*)malloc(size_im);
	//计时
	startTime = clock();
	//设置block，thread数
	dim3 threads(WIDTH);
	dim3 grid(WIDTH-WINSIZE);
	printf("begin gpu computation\n");
	sad_gpu_without_sm<<< grid, threads>>>(d_dis, d_1, d_5, WIDTH);
	hipDeviceSynchronize();
	//停止计时
	endTime = clock();
	double GPUtime = (double)(endTime - startTime) / CLOCKS_PER_SEC;
	printf("GPU:%f\n", GPUtime);
	//把结果拷贝到主机
	hipMemcpy(h_dis, d_dis, size_im, hipMemcpyDeviceToHost);
	//写入文件
        dis = fopen("disp_gpu.txt","w");
        for (i = 0; i < WIDTH*WIDTH; i++) {
	//	if(h_dis[i]>WINSIZE||h_dis[i]<0)
	//		fprintf(dis, "0 ");
	//	else
        		fprintf(dis, "%d " ,h_dis[i]);
        }
        fclose(dis);
	//结束
	free(h_1);
	free(h_5);
	free(h_dis);
	hipFree(d_1);
	hipFree(d_5);
	hipFree(d_dis);

	hipDeviceReset();

	return 0;
}

int** sad(int view1[WIDTH][WIDTH], int view5[WIDTH][WIDTH])
{
		int i, j;	//全图
		int k, l;	//窗口大小
		int n;	//搜索范围
		int result;	//搜索范围内每个窗口的差别
		int min, min_index;		//查找最小值的位置：视差
		int **disp = (int**)malloc(sizeof(int*)*WIDTH);
		for(i = 0; i < WIDTH; i++)
		{
		*(disp+i) = (int*)malloc(sizeof(int)*WIDTH);//开辟列
		}
		for (i = 0; i < WIDTH-WINSIZE; i++) {
			for (j = 0; j < WIDTH-WINSIZE-LIMIT; j++) {
			//对该位置搜索视差
				min = 25500000;
                                min_index = 0;
				for(n = 0; n < LIMIT; n++){
				//每个对应像素求差的绝对值，对整个窗口求和
					result = 0;
					for(k = 0; k<WINSIZE; k++)
						for(l = 0; l<WINSIZE; l++){
					//printf("%d ", view5[i+k][j+l]);	
					result = result + abs(view5[i+k][j+l] - view1[i+k][j+l+n]);
						}
						//printf("%d ",result);
						if(result < min){
							min = result;
							min_index = n;
					//		printf("n: %d, index: %d,re:%d	", n1, min_index, min);
						}
					//	n1++;printf("n1++ ");	//如果更小就更新	
				}
				//printf("%d %d	", min_index, min);
				disp[i][j] = min_index;			
			}
//		printf("roll %d  done\n", i);
		}
		return disp;
}


